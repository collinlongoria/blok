#include "hip/hip_runtime.h"
/*
* File: Renderer_GL
* Project: blok
* Author: Wes Morosan
* Created on: 9/10/2025
* Description: Primarily responsible for raytracing
*/

#include "Cuda_Tracer.hpp"
#include "camera.hpp"
#include "scene.hpp"

#define GLFW_INCLUDE_NONE
#include <glad/glad.h>
#include <GLFW/glfw3.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <stdexcept>
#include <iostream>

using namespace blok;

struct SphereCUDA {
    float3 center;
    float  radius;
    uchar4 color;
};

struct PlaneCUDA {
    float3 normal;
    float  d;
    uchar4 color;
};

struct CameraCUDA {
    float3 pos;
    float3 forward;
    float3 right;
    float3 up;
    float  fovScale;
};

// Utility
__device__ float3 normalize3(const float3& v) {
    float len = sqrtf(v.x*v.x + v.y*v.y + v.z*v.z);
    return make_float3(v.x/len, v.y/len, v.z/len);
}
__device__ float dot3(const float3& a, const float3& b) {
    return a.x*b.x + a.y*b.y + a.z*b.z;
}
__device__ float3 add3(const float3& a, const float3& b) {
    return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}
__device__ float3 sub3(const float3& a, const float3& b) {
    return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}
__device__ float3 mul3(const float3& a, float s) {
    return make_float3(a.x*s, a.y*s, a.z*s);
}

// Intersections
__device__ bool hit_sphere(const SphereCUDA& s, float3 ro, float3 rd, float& tHit) {
    float3 oc = sub3(ro, s.center);
    float a = dot3(rd, rd);
    float b = 2.0f * dot3(oc, rd);
    float c = dot3(oc, oc) - s.radius*s.radius;
    float disc = b*b - 4*a*c;
    if (disc < 0) return false;
    float t0 = (-b - sqrtf(disc)) / (2.0f*a);
    float t1 = (-b + sqrtf(disc)) / (2.0f*a);
    tHit = (t0 > 1e-4f) ? t0 : ((t1 > 1e-4f) ? t1 : -1);
    return tHit > 0;
}

__device__ bool hit_plane(const PlaneCUDA& p, float3 ro, float3 rd, float& tHit) {
    float denom = dot3(p.normal, rd);
    if (fabsf(denom) < 1e-6f) return false;
    float t = -(dot3(p.normal, ro) + p.d) / denom;
    if (t > 1e-4f) { tHit = t; return true; }
    return false;
}

// --- Added: scene helpers for shading, shadows, background, gamma ---
struct HitInfo {
    float  t;
    float3 n;
    uchar4 base;
    bool   hit;
};

__device__ uchar4 lerpU8(uchar4 a, uchar4 b, float t) {
    float it = 1.0f - t;
    return make_uchar4(
        (unsigned char)(a.x*it + b.x*t),
        (unsigned char)(a.y*it + b.y*t),
        (unsigned char)(a.z*it + b.z*t),
        255
    );
}

__device__ HitInfo traceClosest(
    float3 ro, float3 rd,
    SphereCUDA* spheres, int numSpheres,
    PlaneCUDA* planes,  int numPlanes)
{
    HitInfo h; h.t = 1e20f; h.hit = false; h.n = make_float3(0,1,0); h.base = make_uchar4(255,255,255,255);

    // spheres
    for (int i=0; i<numSpheres; ++i) {
        float tHit;
        if (hit_sphere(spheres[i], ro, rd, tHit) && tHit < h.t) {
            h.t = tHit;
            float3 hp = add3(ro, mul3(rd, tHit));
            h.n = normalize3(sub3(hp, spheres[i].center));
            h.base = spheres[i].color;
            h.hit = true;
        }
    }

    // planes (checkerboard)
    for (int i=0; i<numPlanes; ++i) {
        float tHit;
        if (hit_plane(planes[i], ro, rd, tHit) && tHit < h.t) {
            h.t = tHit;
            h.n = normalize3(planes[i].normal);

            // Checker: project hit point onto plane basis
            float3 hp = add3(ro, mul3(rd, tHit));
            // Build a tangent basis for the plane
            float3 n = h.n;
            float3 t = normalize3(fabsf(n.x) > 0.5f ? make_float3(0,1,0) : make_float3(1,0,0));
            t = normalize3(sub3(t, mul3(n, dot3(t,n))));
            float3 b = normalize3(make_float3(
                n.y*t.z - n.z*t.y,
                n.z*t.x - n.x*t.z,
                n.x*t.y - n.y*t.x));

            float u = dot3(hp, t);
            float v = dot3(hp, b);
            int iu = (int)floorf(u);
            int iv = (int)floorf(v);
            bool checker = ((iu + iv) & 1) == 0;

            uchar4 c0 = planes[i].color;              // base color
            uchar4 c1 = make_uchar4(220,220,220,255); // alternate
            h.base = checker ? c0 : c1;
            h.hit = true;
        }
    }

    return h;
}

__device__ bool traceShadow(
    float3 ro, float3 rd, float maxDist,
    SphereCUDA* spheres, int numSpheres,
    PlaneCUDA* planes,  int numPlanes)
{
    float t;
    ro = add3(ro, mul3(rd, 1e-3f));
    for (int i=0; i<numSpheres; ++i) {
        if (hit_sphere(spheres[i], ro, rd, t) && t < maxDist) return true;
    }
    for (int i=0; i<numPlanes; ++i) {
        if (hit_plane(planes[i], ro, rd, t) && t < maxDist) return true;
    }
    return false;
}

__device__ float3 skyGradient(float3 rd) {
    float t = 0.5f*(rd.y + 1.0f);
    float3 top = make_float3(0.5f, 0.7f, 1.0f);
    float3 bot = make_float3(1.0f, 1.0f, 1.0f);
    return add3(mul3(bot, (1.0f - t)), mul3(top, t));
}

__device__ unsigned char toSRGB8(float x) {
    x = fminf(fmaxf(x, 0.0f), 1.0f);
    float g = powf(x, 1.0f/2.2f);
    return (unsigned char)(g * 255.0f + 0.5f);
}

__device__ uchar4 shadePixel(
    float3 ro, float3 rd,
    SphereCUDA* spheres, int numSpheres,
    PlaneCUDA* planes,  int numPlanes,
    float3 lightDir)
{
    HitInfo h = traceClosest(ro, rd, spheres, numSpheres, planes, numPlanes);
    if (!h.hit) {
        float3 c = skyGradient(rd);
        return make_uchar4(toSRGB8(c.x), toSRGB8(c.y), toSRGB8(c.z), 255);
    }

    float3 hp = add3(ro, mul3(rd, h.t));
    float3 n  = h.n;
    float3 l  = normalize3(lightDir);
    float3 v  = normalize3(mul3(rd, -1.0f));
    float3 hlf= normalize3(add3(l, v));

    float ambient = 0.08f;
    float diff    = fmaxf(0.0f, dot3(n, l));
    float spec    = powf(fmaxf(0.0f, dot3(n, hlf)), 48.0f);

    bool shadowed = traceShadow(hp, l, 1e4f, spheres, numSpheres, planes, numPlanes);
    float shadowFactor = shadowed ? 0.25f : 1.0f;

    float3 base = make_float3(h.base.x/255.0f, h.base.y/255.0f, h.base.z/255.0f);

    float3 lit = add3(mul3(base, ambient + diff * shadowFactor),
                      mul3(make_float3(1.0f,1.0f,1.0f), spec * 0.4f * shadowFactor));

    return make_uchar4(toSRGB8(lit.x), toSRGB8(lit.y), toSRGB8(lit.z), 255);
}

// Kernel
__global__ void raytrace_kernel(
    uchar4* pixels, int width, int height,
    CameraCUDA cam,
    SphereCUDA* spheres, int numSpheres,
    PlaneCUDA* planes, int numPlanes,
    float tFrame)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    int idx = y * width + x;

    float u = (2.0f * ((x + 0.5f) / width) - 1.0f) * cam.fovScale;
    float v = (1.0f - 2.0f * ((y + 0.5f) / height)) * cam.fovScale;
    float3 rd = normalize3(add3(add3(cam.forward, mul3(cam.right, u)), mul3(cam.up, v)));
    float3 ro = cam.pos;

    uchar4 color = make_uchar4(100, 149, 237, 255);

    float3 lightDir = normalize3(make_float3(cosf(0.3f + tFrame)*0.6f + 1.0f, 1.0f, -0.5f));

    color = shadePixel(ro, rd, spheres, numSpheres, planes, numPlanes, lightDir);

    pixels[idx] = color;
}

// Helpers
static SphereCUDA toDevice(const Sphere& s) {
    return {
        make_float3(s.center.x, s.center.y, s.center.z),
        s.radius,
        make_uchar4((uint8_t)(s.color.r*255),
                    (uint8_t)(s.color.g*255),
                    (uint8_t)(s.color.b*255),
                    255)
    };
}

static PlaneCUDA toDevice(const Plane& p) {
    return {
        make_float3(p.normal.x, p.normal.y, p.normal.z),
        p.d,
        make_uchar4((uint8_t)(p.color.r*255),
                    (uint8_t)(p.color.g*255),
                    (uint8_t)(p.color.b*255),
                    255)
    };
}

static CameraCUDA toDevice(const Camera& c, int width, int height) {
    float aspect = (float)width / (float)height;
    float fovScale = tanf(0.5f * c.fov * 3.14159f/180.0f);
    return {
        make_float3(c.position.x, c.position.y, c.position.z),
        make_float3(c.forward().x, c.forward().y, c.forward().z),
        make_float3(c.right().x,   c.right().y,   c.right().z),
        make_float3(c.up().x,      c.up().y,      c.up().z),
        fovScale
    };
}

// ---------------- Class ----------------
CudaTracer::CudaTracer(unsigned int width, unsigned int height)
    : m_width(width), m_height(height) {}

CudaTracer::~CudaTracer() {
    cleanup();
}

void CudaTracer::init() {
    glGenBuffers(1, &m_pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, m_pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER,
                 static_cast<GLsizeiptr>(m_width) * m_height * 4,
                 nullptr, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    if (hipGraphicsGLRegisterBuffer(&m_cudaPBO, m_pbo,
                                     cudaGraphicsMapFlagsWriteDiscard) != hipSuccess) {
        throw std::runtime_error("Failed to register PBO with CUDA");
    }

    glGenTextures(1, &m_glTex);
    glBindTexture(GL_TEXTURE_2D, m_glTex);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8,
                 static_cast<GLsizei>(m_width),
                 static_cast<GLsizei>(m_height),
                 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
    glBindTexture(GL_TEXTURE_2D, 0);
}

void CudaTracer::drawFrame(const Camera& cam, const Scene& scene) {
    std::vector<SphereCUDA> spheres;
    for (auto& s : scene.spheres) spheres.push_back(toDevice(s));
    std::vector<PlaneCUDA> planes;
    for (auto& p : scene.planes) planes.push_back(toDevice(p));

    SphereCUDA* dSpheres = nullptr;
    PlaneCUDA* dPlanes   = nullptr;
    if (!spheres.empty())
        hipMalloc(&dSpheres, spheres.size() * sizeof(SphereCUDA));
    if (!planes.empty())
        hipMalloc(&dPlanes, planes.size() * sizeof(PlaneCUDA));

    if (!spheres.empty())
        hipMemcpy(dSpheres, spheres.data(),
                   spheres.size() * sizeof(SphereCUDA),
                   hipMemcpyHostToDevice);
    if (!planes.empty())
        hipMemcpy(dPlanes, planes.data(),
                   planes.size() * sizeof(PlaneCUDA),
                   hipMemcpyHostToDevice);

    CameraCUDA dCam = toDevice(cam, m_width, m_height);

    uchar4* devPtr = nullptr;
    size_t size;
    hipGraphicsMapResources(1, &m_cudaPBO, 0);
    hipGraphicsResourceGetMappedPointer((void**)&devPtr, &size, m_cudaPBO);

    dim3 block(16, 16);
    dim3 grid((m_width + 15) / 16, (m_height + 15) / 16);

    static float t = 0.0f;
    t += 0.01f;

    raytrace_kernel<<<grid, block>>>(devPtr, m_width, m_height, dCam,
                                     dSpheres, (int)spheres.size(),
                                     dPlanes, (int)planes.size(),
                                     t);
    hipDeviceSynchronize();

    hipGraphicsUnmapResources(1, &m_cudaPBO, 0);

    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, m_pbo);
    glBindTexture(GL_TEXTURE_2D, m_glTex);
    glTexSubImage2D(GL_TEXTURE_2D, 0,
                    0, 0,
                    static_cast<GLsizei>(m_width),
                    static_cast<GLsizei>(m_height),
                    GL_RGBA, GL_UNSIGNED_BYTE,
                    nullptr);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    if (dSpheres) hipFree(dSpheres);
    if (dPlanes)  hipFree(dPlanes);
}

void CudaTracer::beginFrame() {}
void CudaTracer::endFrame() {}

void CudaTracer::shutdown() {
    cleanup();
}

void CudaTracer::cleanup() {
    if (m_cudaPBO) {
        hipGraphicsUnregisterResource(m_cudaPBO);
        m_cudaPBO = nullptr;
    }
    if (m_pbo) {
        glDeleteBuffers(1, &m_pbo);
        m_pbo = 0;
    }
    if (m_glTex) {
        glDeleteTextures(1, &m_glTex);
        m_glTex = 0;
    }
}