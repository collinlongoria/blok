#include "hip/hip_runtime.h"
/*
* File: Renderer_GL
* Project: blok
* Author: Wes Morosan
* Created on: 9/10/2025
* Description: Primarily responsible for raytracing
*/

#include "Cuda_Tracer.hpp"

#define GLFW_INCLUDE_NONE
#include <glad/glad.h>
#include <GLFW/glfw3.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <cmath>
#include <stdexcept>
#include <iostream>

using namespace blok;

// fills pixels with a moving gradient
__global__ void raytrace_kernel(uchar4* pixels, int width, int height, float t) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    int idx = y * width + x;

    float u = (float)x / (float)width;
    float v = (float)y / (float)height;

    unsigned char r = static_cast<unsigned char>(u * 255.0f);
    unsigned char g = static_cast<unsigned char>(v * 255.0f);
    unsigned char b = static_cast<unsigned char>((0.5f + 0.5f * sinf(t)) * 255.0f);

    pixels[idx] = make_uchar4(r, g, b, 255);
}

CudaTracer::CudaTracer(unsigned int width, unsigned int height)
    : m_width(width), m_height(height) {}

CudaTracer::~CudaTracer() {
    cleanup();
}

void CudaTracer::init() {

    glGenBuffers(1, &m_pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, m_pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, static_cast<GLsizeiptr>(m_width) * m_height * 4, nullptr, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    {
        hipError_t err = hipGraphicsGLRegisterBuffer(&m_cudaPBO, m_pbo, cudaGraphicsMapFlagsWriteDiscard);
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to register PBO with CUDA");
        }
    }

    glGenTextures(1, &m_glTex);
    glBindTexture(GL_TEXTURE_2D, m_glTex);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST); // no filtering
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8,
                 static_cast<GLsizei>(m_width), static_cast<GLsizei>(m_height),
                 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
    glBindTexture(GL_TEXTURE_2D, 0);
}

void CudaTracer::render() {
    if (!m_cudaPBO || !m_pbo || !m_glTex) return;

    uchar4* devPtr = nullptr;
    size_t mappedSize = 0;
    {
        hipError_t err = hipGraphicsMapResources(1, &m_cudaPBO, 0);
        if (err != hipSuccess) {
            std::cerr << "hipGraphicsMapResources failed: " << static_cast<int>(err) << "\n";
            return;
        }
        err = hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&devPtr), &mappedSize, m_cudaPBO);
        if (err != hipSuccess || !devPtr) {
            std::cerr << "hipGraphicsResourceGetMappedPointer failed\n";
            hipGraphicsUnmapResources(1, &m_cudaPBO, 0);
            return;
        }
    }

    // Launch kernel to fill PBO
    dim3 block(16, 16);
    dim3 grid((m_width + block.x - 1) / block.x,
              (m_height + block.y - 1) / block.y);

    static float t = 0.0f;
    t += 0.02f;

    raytrace_kernel<<<grid, block>>>(devPtr, static_cast<int>(m_width), static_cast<int>(m_height), t);
    hipDeviceSynchronize();

    hipGraphicsUnmapResources(1, &m_cudaPBO, 0);

    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, m_pbo);
    glBindTexture(GL_TEXTURE_2D, m_glTex);
    glTexSubImage2D(GL_TEXTURE_2D, 0,
                    0, 0,
                    static_cast<GLsizei>(m_width), static_cast<GLsizei>(m_height),
                    GL_RGBA, GL_UNSIGNED_BYTE,
                    nullptr);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
    glBindTexture(GL_TEXTURE_2D, 0);
}

void CudaTracer::cleanup() {
    if (m_cudaPBO) {
        hipGraphicsUnregisterResource(m_cudaPBO);
        m_cudaPBO = nullptr;
    }
    if (m_pbo) {
        glDeleteBuffers(1, &m_pbo);
        m_pbo = 0;
    }
    if (m_glTex) {
        glDeleteTextures(1, &m_glTex);
        m_glTex = 0;
    }
}
